#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N 10000000

__global__ void nonlinear_kernel(float* x, float* y, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        y[idx] = sinf(x[idx]) + logf(x[idx] + 1.0f) + sqrtf(x[idx]);
    }
}

int main() {
    float *x_host, *y_host;
    float *x_dev, *y_dev;

    x_host = (float*)malloc(N * sizeof(float));
    y_host = (float*)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x_host[i] = ((float)rand() / RAND_MAX);
    }

    hipMalloc(&x_dev, N * sizeof(float));
    hipMalloc(&y_dev, N * sizeof(float));
    hipMemcpy(x_dev, x_host, N * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    printf("Threads per block: %d\n", threadsPerBlock);
    printf("Blocks per grid:   %d\n", blocksPerGrid);
    printf("Total GPU threads launched: %d\n", blocksPerGrid * threadsPerBlock);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    nonlinear_kernel<<<blocksPerGrid, threadsPerBlock>>>(x_dev, y_dev, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("GPU time: %.4f s\n", ms / 1000.0f);

    hipMemcpy(y_host, y_dev, N * sizeof(float), hipMemcpyDeviceToHost);

    free(x_host);
    free(y_host);
    hipFree(x_dev);
    hipFree(y_dev);
    return 0;
}